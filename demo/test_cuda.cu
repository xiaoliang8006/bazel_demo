#include <iostream>
#include <hip/hip_runtime.h>

// 内核函数：向量加法
__global__ void vectorAdd(float *A, float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

// 内核函数：初始化向量数据
__global__ void initializeVectors(float *A, float *B, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        A[i] = static_cast<float>(i);
        B[i] = static_cast<float>(i * 2);
    }
}

int main(void) {
    int numElements = 50000;
    int threadsPerBlock = 256;
    int numBlocks = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    size_t size = numElements * sizeof(float);

    // 分配主机内存
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // 初始化向量数据
    initializeVectors<<<numBlocks, threadsPerBlock>>>(h_A, h_B, numElements);
    hipDeviceSynchronize();

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 执行向量加法

    vectorAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipDeviceSynchronize();

    // 将结果从设备复制回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 打印结果
    for (int i = 0; i < numElements; ++i) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    }

    // 释放分配的内存
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}